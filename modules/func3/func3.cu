#include <hip/hip_runtime.h>

#include "func1/func1.cuh"
#include "func2/func2.cuh"
#include "func3.cuh"

namespace test
{
namespace func3
{

__device__ __host__ inline  float AddThree(float x)
{
    return func1::AddOne(x) + func2::AddTwo(x);
}

} // namespace func3
} // namespace test
