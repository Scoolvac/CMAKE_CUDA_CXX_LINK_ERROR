#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "func3/func3.cuh"
#include "func4.cuh"

#define N 256

namespace test
{
namespace func4
{

__global__ void AddThreeCUDA(float* arr)
{
    const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N)
    {
        arr[idx] = func3::AddThree(arr[idx]);
    }
}
    
} // namespace func4
}
