#include <hip/hip_runtime.h>

#include "func1.cuh"



namespace test
{
namespace func1
{

__device__ __host__  float AddOne(float x)
{
    return x + 1.f;
}

} // namespace func1
} // namespace test
