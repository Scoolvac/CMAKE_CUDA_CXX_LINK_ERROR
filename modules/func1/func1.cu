#include <hip/hip_runtime.h>

#include "func1.cuh"



namespace test
{
namespace func1
{

__device__ __host__ inline  float AddOne(float x)
{
    return x + 1.f;
}

} // namespace func1
} // namespace test
