#include <hip/hip_runtime.h>

#include "func1/func1.cuh"
#include "func2.cuh"


namespace test
{
namespace func2
{

__device__ __host__ inline  float AddTwo(float x)
{
    return func1::AddOne(x) + func1::AddOne(x);
}

} // namespace func2
} // namespace test
