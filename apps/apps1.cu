#include "hip/hip_runtime.h"
#define N 256

#include <hip/hip_runtime.h>

#include "func1/func1.cuh"
#include "func2/func2.cuh"
#include "func3/func3.cuh"

__global__ void AddThree(float* arr)
{
    const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N)
    {
        arr[idx] = test::func3::AddThree(arr[idx]);
    }
}

int main()
{
    float* arr = new float[N];
    
    for (unsigned int i = 0; i < N; ++i)
    {
        arr[i] = test::func3::AddThree(0.f);
    }

    hipMallocManaged(&arr, sizeof(float) * N);

    AddThree<<<N,1>>>(arr);
}